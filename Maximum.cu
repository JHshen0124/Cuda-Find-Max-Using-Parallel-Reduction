#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct Startup{
    int random_range = 100;
    int threads_per_block = 1024;
} startup;

struct DataSet{
    float* values;
    int  size;
};

/*
DataSet* createDeviceDataset(DataSet host){
    DataSet* device_dataset = (DataSet*)malloc(sizeof(DataSet));
    gpuErrchk(cudaMalloc((void **)&host_copy,  sizeof(DataSet)));
    gpuErrchk(cudaMalloc((void **)&device_dataset,  sizeof(DataSet)));
    gpuErrchk(cudaMemcpy(device_d, input.values, sizeOfDataSet(input) , cudaMemcpyHostToDevice));
    return device_dataset;

}*/

DataSet generateRandomDataSet(int size){
    DataSet data;
    data.size = size;
    data.values = (float*)malloc(sizeof(float)*data.size);

    for (int i = 0; i < data.size; i++)
        data.values[i] = (float)(rand()%startup.random_range);

    return data;
}

__global__ void MaxValue_1(float* data, int data_size){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < data_size){
        for(int stride=1; stride < data_size; stride *= 2) {
            if (idx % (2*stride) == 0) {
                float lhs = data[idx];
                float rhs = data[idx + stride];
                data[idx] = lhs < rhs ? rhs : lhs;
            }
            __syncthreads();
        }
    }
}

float calculateMaxValue(DataSet data){
    float* device_data;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);    

    gpuErrchk(hipMalloc((void **)&device_data,  sizeof(float)*data.size));
    gpuErrchk(hipMemcpy(device_data, data.values, sizeof(float)*data.size, hipMemcpyHostToDevice));


    int threads_needed = data.size;
    hipEventRecord(start);
    MaxValue_1<<< threads_needed/ startup.threads_per_block + 1, startup.threads_per_block >>>(device_data, data.size);
    hipEventRecord(stop);
    gpuErrchk(hipGetLastError());
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel Executed in %.6g\n", milliseconds);

    float max_value;
    gpuErrchk(hipMemcpy(&max_value, device_data, sizeof(float), hipMemcpyDeviceToHost));
    return max_value;
}

void printDataSet(DataSet data){
    for (int i = 0; i < data.size; i++)
        printf("%.6g, ", data.values[i]);
    printf("\n");
}

 


int main(int argc, char** argv){
    srand(time(nullptr));
    DataSet random = generateRandomDataSet(10);
    printDataSet(random);
    float max = calculateMaxValue(random);
    printf("The maximum value is: %g", max);
}