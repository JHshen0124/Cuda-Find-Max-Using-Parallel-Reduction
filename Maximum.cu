#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct Startup{
    int random_range = 50;
    int threads_per_block = 1024;
} startup;

struct DataSet{
    float* values;
    int  size;
};

/*
DataSet* createDeviceDataset(DataSet host){
    DataSet* device_dataset = (DataSet*)malloc(sizeof(DataSet));
    gpuErrchk(cudaMalloc((void **)&host_copy,  sizeof(DataSet)));
    gpuErrchk(cudaMalloc((void **)&device_dataset,  sizeof(DataSet)));
    gpuErrchk(cudaMemcpy(device_d, input.values, sizeOfDataSet(input) , cudaMemcpyHostToDevice));
    return device_dataset;

}*/

DataSet generateRandomDataSet(int size){
    DataSet data;
    data.size = size;
    data.values = (float*)malloc(sizeof(float)*data.size);

    for (int i = 0; i < data.size; i++)
        data.values[i] = (float)(rand()%startup.random_range);

    return data;
}

__global__ void MaxValue_1(float* data, int data_size){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < data_size){
        true;
    }
    //input[idx] > input[idx+1]
}

float calculateMaxValue(DataSet data){
    float* device_data;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);    

    gpuErrchk(hipMalloc((void **)&device_data,  sizeof(float)*data.size));
    gpuErrchk(hipMalloc((void **)&data.values,  sizeof(float)*data.size));

    int threads_needed = data.size;
    hipEventRecord(start);
    MaxValue_1<<< threads_needed/ startup.threads_per_block + 1, startup.threads_per_block >>>(device_data, data.size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel Executed in %.6g\n", milliseconds);
    return milliseconds;
}

void printDataSet(DataSet data){
    for (int i = 0; i < data.size; i++)
        printf("%.6g, ", data.values[i]);
    printf("\n");
}

 


int main(int argc, char** argv){
    DataSet random = generateRandomDataSet(10000);
    float max = calculateMaxValue(random);
    //printDataSet(random);
}