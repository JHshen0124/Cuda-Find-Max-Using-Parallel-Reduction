#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct Startup{
    int random_range = 50;
    int threads_per_block = 1024;
} startup;

struct DataSet{
    float* values;
    int  size;
};

DataSet* createDeviceDataset(DataSet host){
    DataSet host_copy = {nullptr, host.size};
    gpuErrchk(hipMalloc((void **)&host_copy.values,  host.size*sizeof(float));
    gpuErrchk(hipMalloc((void **)&host_copy,  host.size*sizeof(float));

    DataSet* device_dataset = (DataSet*)malloc(sizeof(DataSet));
    gpuErrchk(hipMalloc((void **)&host_copy,  sizeof(DataSet));
    gpuErrchk(hipMalloc((void **)&device_dataset,  sizeof(DataSet));
    gpuErrchk(hipMemcpy(device_d, input.values, sizeOfDataSet(input) , hipMemcpyHostToDevice));
    return device_dataset;

}

DataSet generateRandomDataSet(int size){
    DataSet data;
    data.size = size;
    data.values = (float*)malloc(sizeof(float)*data.size);

    for (int i = 0; i < data.size; i++)
        data.values[i] = (float)(rand()%startup.random_range);

    return data;
}

__global__ void MaxValue_1(DataSet data){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    //input[idx] > input[idx+1]
}

float calculateMaxValue(DataSet data){
    float* device_data;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);    

    int threads_needed = data.size;
    hipEventRecord(start);
    MaxValue_1<<< threads_needed/ startup.threads_per_block + 1, startup.threads_per_block >>>(device_data);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel Executed in %.6g\n", milliseconds);
    return milliseconds;
}

void printDataSet(DataSet data){
    for (int i = 0; i < data.size; i++)
        printf("%.6g, ", data.values[i]);
    printf("\n");
}

 


int main(int argc, char** argv){
    DataSet random = generateRandomDataSet(10000);
    float max = calculateMaxValue(random);
    //printDataSet(random);
}